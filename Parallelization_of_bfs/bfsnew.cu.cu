#include "hip/hip_runtime.h"


//#include <device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define NUM_NODES 10

typedef struct
{
	int start;     // Index of first adjacent node in Ea	
	int length;    // Number of adjacent nodes 
} Node;

__global__ void CUDA_BFS_KERNEL(Node *Va, int *Ea, bool *Fa, bool *Xa, int *Ca,bool *done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		*done = false;
		//printf("\n id is %d ", id);
		printf("\n Fa[id] is %d --> Xa[id] is %d", Fa[id], Xa[id]);

	if (Fa[id] == true && Xa[id] == false)
	{
		printf("\n id--> %d ", id); //This printf gives the order of vertices in BFS
		//printf("\n Fa[id] is %d", Fa[id]);
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads(); 
		//int k = 0;
		//int i;
		int start = Va[id].start;
		//printf("\n Va[id] is %d", Va[id]);

		int end = start + Va[id].length;
		for (int i = start; i < end; i++) 
		{
			int nid = Ea[i];
			//printf("\n value of i is %d", i);
			//printf("\n nid is %d", nid); 
			if (Xa[nid] == false)
			{
				//printf("\n Va[id] is %d", Va[id]);
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}

		}

	}

}

// The BFS frontier corresponds to all the nodes being processed at the current level.


int main()
{




	 Node node[NUM_NODES];
	
	
	//int edgesSize = 2 * NUM_NODES;
	int edges[NUM_NODES];

	for(int i = 0; i<10; i = i+5){
		node[i+0].start = i+0;
		node[i+0].length = 2;

		node[i+1].start = i+2;
		node[i+1].length = 1;

		node[i+2].start = i+3;
		node[i+2].length = 1;

		node[i+3].start = i+4;
		node[i+3].length = 1;

		node[i+4].start = i+5;
		if(i == 5)
		node[i+4].length = 0;
		else
		node[i+4].length = 1;

		edges[i+0] = i+1;
		edges[i+1] = i+2;	
		edges[i+2] = i+4;
		edges[i+3] = i+3;
		if(i == 5)
		edges[i+4] = i+4;
		else
		edges[i+4] = i+5;
	}
	//for(int i = 0; i< 10; i++){
	//	printf("\n node of %d is %d", i, node[i]);
	//	printf("\n edge of %d is %d", i, edges[i]);
	//}
	bool frontier[NUM_NODES] = { false };
	bool visited[NUM_NODES] = { false };
	int cost[NUM_NODES] = { 0 };

	int source = 0;
	frontier[source] = true;

	Node* Va;
	hipMalloc((void**)&Va, sizeof(Node)*NUM_NODES);
	hipMemcpy(Va, node, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ea;
	hipMalloc((void**)&Ea, sizeof(Node)*NUM_NODES);
	hipMemcpy(Ea, edges, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Fa;
	hipMalloc((void**)&Fa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Fa, frontier, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Xa;
	hipMalloc((void**)&Xa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Xa, visited, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ca;
	hipMalloc((void**)&Ca, sizeof(int)*NUM_NODES);
	hipMemcpy(Ca, cost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	

	int num_blks = 1;
	int threads = 10;



	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: ");
	do {
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL <<<num_blks, threads >>>(Va, Ea, Fa, Xa, Ca,d_done);
		hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost);

	} while (!done);




	hipMemcpy(cost, Ca, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost);
	
	printf("Number of times the kernel is called : %d \n", count);


	printf("\nCost: ");
	for (int i = 0; i<NUM_NODES; i++)
		printf( "%d    ", cost[i]);
	printf("\n");
	//_getch();
	
}